#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "object.cuh"
#include <cstdlib>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include "index_.h"
#include "cuda_guard.cuh"

Object Objects::objects_[Objects::kTotalObjectNum];

__global__ void devStep(hiprandState* s, Object* objects_, const G_Grid::Cell* __restrict__ grid_) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	auto& o = objects_[idx];
	o.position_ += o.speed_;

	auto& edge = grid_[o.cell_id].vertex_[o.vertex_pos_].edges_[o.edge_pos_];
	auto edge_length = edge.length_;
	while (o.position_ >= edge_length) {
		o.position_ -= edge_length;
		o.cell_id = edge.to_cell_;
		o.vertex_pos_ = edge.to_vertex_pos_;
		int new_edge_pos = hiprand_uniform(s + idx) * grid_[o.cell_id].vertex_[o.vertex_pos_].edge_num_;
		auto& new_edge = grid_[o.cell_id].vertex_[o.vertex_pos_].edges_[new_edge_pos];
		o.edge_pos_ = new_edge_pos;
		o.edge_id_ = new_edge.id_;
	}
}

void Objects::Step() {
	int size = Objects::kTotalObjectNum;

	devStep<<<16, size / 16>>>(CudaGuard::pd_curand_state_, CudaGuard::pd_objects_, CudaGuard::pd_grid_);
	hipDeviceSynchronize();
	hipMemcpy(Objects::objects_, CudaGuard::pd_objects_, size * sizeof(Object), hipMemcpyDeviceToHost);
}

__global__ void devInitialize(hiprandState* s, Object* objects, const G_Grid::Cell* __restrict__ grid_) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	hiprand_init(1232, idx, 0, &s[idx]);

	objects[idx].id_ = idx;
	objects[idx].cell_id = hiprand_uniform(s + idx) * G_Grid::kCellNum;

	objects[idx].vertex_pos_ = hiprand_uniform(s + idx) * grid_[objects[idx].cell_id].vertex_num;
	auto& vertex = grid_[objects[idx].cell_id].vertex_[objects[idx].vertex_pos_];

	objects[idx].edge_pos_ = hiprand_uniform(s + idx) * vertex.edge_num_;
	auto& edge = vertex.edges_[objects[idx].edge_pos_];
	objects[idx].edge_id_ = edge.id_;

	objects[idx].position_ = hiprand_uniform(s + idx) * edge.length_;
	objects[idx].speed_ = Objects::kMinSpeed + hiprand_uniform(s + idx) * (Objects::kMaxSpeed - Objects::kMinSpeed);
}

void Objects::Initialize() {
	memset(Objects::objects_, 0, sizeof(Object) * kTotalObjectNum);

	int size = Objects::kTotalObjectNum;

	devInitialize<<<16, size / 16>>>(CudaGuard::pd_curand_state_, CudaGuard::pd_objects_, CudaGuard::pd_grid_);
	hipDeviceSynchronize();
	hipMemcpy(Objects::objects_, CudaGuard::pd_objects_, size * sizeof(Object), hipMemcpyDeviceToHost);
}
