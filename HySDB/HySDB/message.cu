#include "hip/hip_runtime.h"
#include "stdafx.h"

#include "hip/hip_runtime.h"
#include ""

#ifndef __HIPCC__
#define __HIPCC__
#endif

#ifndef __THROW
#define __THROW
#endif

#include <hip/device_functions.h>
#include "message.h"
#include "algorithm"
#include "now.h"

__global__ void xsfl_msg_knl(int t, int* o_num, Message* A, Message* T) {
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	int bundle_id = id / MessageBucket::kRhoB;

	Message cached_messages[5];

	for (int i = id * MessageBucket::kRhoB; i < MessageBucket::kRhoB; ++i) {
		Message m = A[i];
		for (int j = 4; j >= 0; --j) {
			for (int k = 0; k < 5; ++k) {
				if (cached_messages[k].oid == m.oid) {
					if (cached_messages[k].t < m.t) {
						cached_messages[k] = m;
					}
					break;
				}
				if(cached_messages[k].oid == 0) {
					cached_messages[k] = m;
					break;
				}
			}				

			m.oid = __shfl_xor_sync(0xFFFFFFFF, m.oid, 1 << j);
//			m.eid = __shfl_xor(m.eid, 1 << j);
//			m.rho = __shfl_xor(m.rho, 1 << j);
//			m.t = __shfl_xor(m.t, 1 << j);
		}
		for (int j = 0; j < 4; ++j) {
			T[bundle_id] = m;
		}
	}
}

MessageBucket* MessageLists::lists_[Index::kCellNum];

void MessageLists::MessageCleaning(std::vector<int> lists, int message_out_num, Message* messages) {
	int n_to_clean = std::accumulate(lists.begin(), lists.end(), 0,
	                                 [](int t, int i) ->int {
	                                 auto pm = lists_[i];
	                                 int total = 0;
	                                 while (pm != nullptr && (Now::now() - pm->t < kTimeDelta)) {
		                                 //TODO delete if obsolete
		                                 total++;
		                                 pm = pm->p;
	                                 }
	                                 return t + total;
                                 });

	Message *h_buckets, *d_buckets, *d_T;
	int n_message_out, *d_m;
	hipHostMalloc(&h_buckets, sizeof(Message) * MessageBucket::kRhoB * n_to_clean);
	hipMalloc(&d_buckets, MessageBucket::kRhoB * n_to_clean);
	hipMalloc(&d_m, sizeof(int));
	hipMalloc(&d_T, sizeof(Message) * n_to_clean);
	hipMemcpy(d_buckets, h_buckets, sizeof(Message) * MessageBucket::kRhoB * n_to_clean, hipMemcpyHostToDevice);

	hipSetDevice(0);

	dim3 block(128);
	dim3 grid(n_to_clean / 128);

	//	xsfl_msg_knl << <grid, block, 0, CudaStreamControler::getStream()>> >(Now::now(), d_m, d_buckets, d_T);

	hipDeviceSynchronize();

	hipMemcpy(h_buckets, d_buckets, sizeof(Message) * MessageBucket::kRhoB * n_to_clean, hipMemcpyDeviceToHost);
	hipMemcpy(d_m, &n_message_out, sizeof(int), hipMemcpyDeviceToHost);

	hipFree(d_buckets);
	hipFree(d_m);
	hipFree(d_T);
	hipHostFree(h_buckets);
}
