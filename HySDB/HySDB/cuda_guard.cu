#include "stdafx.h"
#include "cuda_guard.cuh"
#include <hip/hip_runtime_api.h>
#include ""
#include "index_.h"

G_Grid::Cell* CudaGuard::pd_grid_;
Object* CudaGuard::pd_objects_;
hiprandState* CudaGuard::pd_curand_state_;

CudaGuard::CudaGuard() {
	hipSetDevice(0);

	G_Grid::Generate_Randomly();
	hipMalloc((void**)&pd_grid_, sizeof(G_Grid::grid_));
	hipMemcpy(pd_grid_, G_Grid::grid_, sizeof(G_Grid::grid_), hipMemcpyHostToDevice);

	hipMalloc((void**)&pd_objects_, Objects::kTotalObjectNum * sizeof(Object));
	hipMemset(pd_objects_, 0, Objects::kTotalObjectNum * sizeof(Object));

	hipMalloc((void**)&pd_curand_state_, Objects::kTotalObjectNum * sizeof(hiprandState));
}

CudaGuard::~CudaGuard() {
	hipFree(pd_objects_);
	hipFree(pd_grid_);
	hipFree(pd_curand_state_);

	hipDeviceReset();
}
