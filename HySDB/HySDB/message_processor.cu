#include "hip/hip_runtime.h"
#include "stdafx.h"
#include "hip/hip_runtime.h"
#include ""

#ifndef __HIPCC__
#define __HIPCC__
#endif

#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>

#include "index_.h"
#include "object.cuh"
#include "message_processor.cuh"
#include "now.h"
#include "cuda_stream_controller.h"

__global__ void xsfl_msg_knl(int n, MessageC* A, MessageC* T) {
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	int bundle_id = id / MessageBucket::kRhoB;

	MessageC cached_messages[5];

	for (int i = id * MessageBucket::kRhoB; i < MessageBucket::kRhoB; ++i) {
		MessageC m = A[i];
		for (int j = 4; j >= 0; --j) {
			for (int k = 0; k < 5; ++k) {
				if (cached_messages[k].oid == m.oid) {
					if (cached_messages[k].t < m.t) {
						cached_messages[k] = m;
					}
					break;
				}
				if (cached_messages[k].oid == 0) {
					cached_messages[k] = m;
					break;
				}
			}

			m.oid = __shfl_xor_sync(0xFFFFFFFF, m.oid, 1 << j);
			m.eid = __shfl_xor_sync(0xFFFFFFFF, m.eid, 1 << j);
			m.rho = __shfl_xor_sync(0xFFFFFFFF, m.rho, 1 << j);
			m.t = __shfl_xor_sync(0xFFFFFFFF, m.t, 1 << j);
		}
		for (int j = 0; j < 4; ++j) {
			T[m.oid * n + bundle_id] = m;
		}
	}
}

__global__ void clct_knl(int n, int* o_num, MessageC *R, MessageC* T) {
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	int bundle_id = id / MessageBucket::kRhoB;

	MessageC m;
	for (int i = id * n; i < id * n + n; ++i) {
		if (T[i].oid == id && T[i].t > m.t) {
			m = T[i];
		}
		if (m.oid != 0) {
			for (int j = m.cid * G_Grid::kMaxObjectsPerCell; j < m.cid * G_Grid::kMaxObjectsPerCell + G_Grid::kMaxObjectsPerCell ; ++j) {
				if (R[j].oid == 0) R[j] = m;
			}
		}
	}
}

MessageBucket* MessageLists::lists_[G_Grid::kCellNum];

void MessageLists::MessageCleaning(std::vector<int> lists, int message_out_num, MessageC* messages) {
	int n_to_clean = std::accumulate(lists.begin(), lists.end(), 0,
	                                 [](int t, int i) -> int {
	                                 auto pm = lists_[i];
	                                 int total = 0;
	                                 while (pm != nullptr && (Now::now() - pm->t < kTimeDelta)) {
		                                 //TODO delete if obsolete
		                                 total++;
		                                 pm = pm->p;
	                                 }
	                                 return t + total;
                                 });

	MessageC *h_buckets, *d_buckets, *d_T, *d_R, *h_R;
	int n_message_out, *d_m;
	hipHostMalloc(&h_buckets, sizeof(MessageC) * MessageBucket::kRhoB * n_to_clean);
	hipMalloc(&d_buckets, MessageBucket::kRhoB * n_to_clean);
	hipMalloc(&d_m, sizeof(int));
	hipMalloc(&d_T, sizeof(MessageC) * Objects::kTotalObjectNum * n_to_clean);
	hipMalloc(&d_R, sizeof(MessageC) * G_Grid::kCellNum * G_Grid::kMaxObjectsPerCell);
	hipHostMalloc(&h_R, sizeof(MessageC) * G_Grid::kCellNum * G_Grid::kMaxObjectsPerCell);
	hipMemcpy(d_buckets, h_buckets, sizeof(MessageC) * MessageBucket::kRhoB * n_to_clean, hipMemcpyHostToDevice);

	hipSetDevice(0);

	dim3 block(128);
	dim3 grid(n_to_clean / 128);

	xsfl_msg_knl << <grid, block, 0, CudaStreamControler::getStream()>> >(n_to_clean, d_buckets, d_T);

	clct_knl << <dim3(Objects::kTotalObjectNum / 128), block, 0, CudaStreamControler::getStream()>> >(n_to_clean, d_m, d_R, d_T);

	hipDeviceSynchronize();

	hipMemcpy(d_R, h_R, sizeof(MessageC) * MessageBucket::kRhoB * n_to_clean, hipMemcpyDeviceToHost);
	hipMemcpy(d_m, &n_message_out, sizeof(int), hipMemcpyDeviceToHost);

	hipFree(d_buckets);
	hipFree(d_m);
	hipFree(d_T);
	hipFree(d_R);
	hipHostFree(h_buckets);
	hipHostFree(h_R);
}
