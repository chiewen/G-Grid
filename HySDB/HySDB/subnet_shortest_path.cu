#include "hip/hip_runtime.h"
#include "stdafx.h"
#include "hip/hip_runtime.h"
#include ""

#ifndef __HIPCC__
#define __HIPCC__
#endif

#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>

#include "cuda_guard.cuh"
#include "subnet_shortest_path.cuh"
#include "index_.h"

void SubnetBF::handle_unresolved(int* cells, int cell_num, int start_cell, int start_vertex, int start_edge, int distance, SBfResult* result, int edge_num) {
	int grid_to_result[G_Grid::kCellNum];
	for (int i = 0; i < G_Grid::kCellNum; ++i) {
		grid_to_result[i] = -1;
	}

	int result_idx = 0;
	for (int i = 0; i < cell_num; ++i) {
		grid_to_result[cells[i]] = i;
		for (int j = 0; j < G_Grid::kMaxVerticesPerCell; ++j) {
			auto& r = result[result_idx++];
			r.id_ = G_Grid::grid_[cells[i]].vertex_[j].id_;
			r.distance_ = std::numeric_limits<int>::max();
			r.previous_vertex_id = 0;
		}
	}
	auto& edge = G_Grid::grid_[start_cell].vertex_[start_vertex].edges_[start_edge];
	result[G_Grid::kMaxVerticesPerCell * grid_to_result[edge.to_cell_] + edge.to_vertex_pos_].distance_ = edge.length_ - distance;
	for (int i = 0; i < edge_num; ++i) {
		for (int j = 0; j < cell_num; ++j) {
			auto& c = G_Grid::grid_[cells[j]];
			for (int k = 0; k < G_Grid::kMaxVerticesPerCell; ++k) {
				auto& v = c.vertex_[k];
				int v_dist = result[G_Grid::kMaxVerticesPerCell * grid_to_result[cells[j]] + k].distance_;
				if (v_dist < std::numeric_limits<int>::max()) {
					for (int l = 0; l < G_Grid::kMaxEdgesPerVertex; ++l) {
						auto& e = v.edges_[l];
						if (e.id_ != 0 && grid_to_result[e.to_cell_] != -1) {
							auto& r = result[G_Grid::kMaxVerticesPerCell * grid_to_result[e.to_cell_] + e.to_vertex_pos_];
							if (r.distance_ > v_dist + e.length_) {
								r.distance_ = v_dist + e.length_;
								r.previous_vertex_id = v.id_;
							}
						}
					}
				}
			}
		}
	}
}

__global__ void fst_k_knl(int* cells, int cell_num, int * grid_to_result, SubnetBF::SBfResult* result, G_Grid::Cell* grid_, int edge_num, int int_max) {
	int cells_per_loop = SubnetBF::kMaxThreadsPerBlock / G_Grid::kMaxVerticesPerCell;
	int loop_num = (cell_num + cells_per_loop - 1) / cells_per_loop;

	for (int i = 0; i < edge_num; ++i) {
		for (int j = 0; j < loop_num; ++j) {
			int current_cell = threadIdx.y + cells_per_loop * j;
			if (current_cell < cell_num) {
				auto& cell = grid_[cells[current_cell]];
				auto& vertex = cell.vertex_[threadIdx.x];
				int v_dist = result[G_Grid::kMaxVerticesPerCell * grid_to_result[cells[current_cell]] + threadIdx.x].distance_;
				if (v_dist < int_max) {
					for (int l = 0; l < G_Grid::kMaxEdgesPerVertex; ++l) {
						auto& e = vertex.edges_[l];
						if (e.id_ != 0 && grid_to_result[e.to_cell_] != -1) {
							auto& r = result[G_Grid::kMaxVerticesPerCell * grid_to_result[e.to_cell_] + e.to_vertex_pos_];
							if (r.distance_ > v_dist + e.length_) {
								r.distance_ = v_dist + e.length_;
								r.previous_vertex_id = vertex.id_;
							}
						}
					}
				}
			}
		}
		__syncthreads();
	}
}

void SubnetBF::find_first_k(int* cells, int cell_num, int start_cell, int start_vertex, int start_edge, int distance, SBfResult* result, int edge_num) {
	int grid_to_result[G_Grid::kCellNum];
	for (int i = 0; i < G_Grid::kCellNum; ++i) {
		grid_to_result[i] = -1;
	}

	int result_idx = 0;
	for (int i = 0; i < cell_num; ++i) {
		grid_to_result[cells[i]] = i;
		for (int j = 0; j < G_Grid::kMaxVerticesPerCell; ++j) {
			auto& r = result[result_idx++];
			r.id_ = G_Grid::grid_[cells[i]].vertex_[j].id_;
			r.distance_ = std::numeric_limits<int>::max();
			r.previous_vertex_id = 0;
		}
	}

	auto& edge = G_Grid::grid_[start_cell].vertex_[start_vertex].edges_[start_edge];
	result[G_Grid::kMaxVerticesPerCell * grid_to_result[edge.to_cell_] + edge.to_vertex_pos_].distance_ = edge.length_ - distance;
	
	int * d_cells;
	hipMalloc(&d_cells, sizeof(int) * cell_num);
	hipMemcpy(d_cells, cells, sizeof(int) * cell_num, hipMemcpyHostToDevice);

	int * d_grid_to_result;
	hipMalloc(&d_grid_to_result, sizeof(int) * G_Grid::kCellNum);
	hipMemcpy(d_grid_to_result, grid_to_result, sizeof(int) * G_Grid::kCellNum, hipMemcpyHostToDevice);

	SBfResult * d_result;
	hipMalloc(&d_result, sizeof(SBfResult) * cell_num * G_Grid::kMaxVerticesPerCell);
	hipMemcpy(d_result, result, sizeof(SBfResult) * cell_num * G_Grid::kMaxVerticesPerCell, hipMemcpyHostToDevice);

	dim3 block(G_Grid::kMaxVerticesPerCell, kMaxThreadsPerBlock / G_Grid::kMaxVerticesPerCell);
	fst_k_knl <<<1, block>>>(d_cells, cell_num, d_grid_to_result, d_result, CudaGuard::pd_grid_, edge_num, std::numeric_limits<int>::max());

	hipMemcpy(result, d_result, sizeof(SBfResult) * cell_num * G_Grid::kMaxVerticesPerCell, hipMemcpyDeviceToHost);

	hipFree(d_grid_to_result);
	hipFree(d_result);
	hipFree(d_cells);
}
